
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

__global__ void kernel(float* r, float* g, float* b, float* r2, float* g2, float* b2, int N, int M, int x) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;

	if(tId < N * M) {
    int c = tId % N;
    int des = ( ((c / x ) % 2 != 0)) * (tId - x) + ((c / x ) % 2 == 0) * (tId + x);
    int res = ( des <= N * M ) * des;
		r2[res] = r[tId];
		g2[res] = g[tId];
		b2[res] = b[tId];
	}
}

int main(int argc, char const *argv[]) {
	hipEvent_t ct1, ct2;
	float *R, *G, *B;
	float *R_gpu, *G_gpu, *B_gpu;
	float *R_gpu2, *G_gpu2, *B_gpu2;
	float dt;

	int M, N, X = atoi(argv[3]);
	
	FILE *in = fopen(argv[1], "r");
	fscanf(in, "%d %d", &M, &N);

	int block_size = 256;
	int grid_size = (int) ceil( (float) M * N / block_size);

	R = new float[M*N];
	G = new float[M*N];
	B = new float[M*N];

	hipMalloc(&R_gpu, sizeof(float) * N * M);
	hipMalloc(&G_gpu, sizeof(float) * N * M);
	hipMalloc(&B_gpu, sizeof(float) * N * M);

	hipMalloc(&R_gpu2, sizeof(float) * N * M);
	hipMalloc(&G_gpu2, sizeof(float) * N * M);
	hipMalloc(&B_gpu2, sizeof(float) * N * M);

	for (int i = 0; i < N * M; ++i) fscanf (in, "%f", &R[i]);
	for (int i = 0; i < N * M; ++i) fscanf (in, "%f", &G[i]);
	for (int i = 0; i < N * M; ++i) fscanf (in, "%f", &B[i]);
	
	fclose (in);

	hipMemcpy(R_gpu, R, sizeof(float) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(G_gpu, G, sizeof(float) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(float) * N * M, hipMemcpyHostToDevice);
    
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernel<<<grid_size, block_size>>>(R_gpu, G_gpu, B_gpu, R_gpu2, G_gpu2, B_gpu2, N, M, X);

	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	cout << "Tiempo: " << dt << "[ms]" << '\n';

	hipMemcpy(R, R_gpu2, sizeof(float) * N * M, hipMemcpyDeviceToHost);
	hipMemcpy(G, G_gpu2, sizeof(float) * N * M, hipMemcpyDeviceToHost);
	hipMemcpy(B, B_gpu2, sizeof(float) * N * M, hipMemcpyDeviceToHost);

	FILE * res;
	res = fopen (argv[2],"w");
	fprintf(res, "%d %d\n", M, N);

	for (int i = 0; i < N * M; ++i) fprintf(res, "%f%c", R[i], i == N * M - 1 ? '\n' : ' ');
	for (int i = 0; i < N * M; ++i) fprintf(res, "%f%c", G[i], i == N * M - 1 ? '\n' : ' ');
	for (int i = 0; i < N * M; ++i) fprintf(res, "%f%c", B[i], i == N * M - 1 ? '\n' : ' ');

	fclose (res);

	hipFree(R_gpu);
	hipFree(G_gpu);
	hipFree(B_gpu);

	hipFree(R_gpu2);
	hipFree(G_gpu2);
	hipFree(B_gpu2);

	delete R;
	delete G;
	delete B;

	return 0;
}